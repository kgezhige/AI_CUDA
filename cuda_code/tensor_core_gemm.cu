#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>
#include <iostream>

// 检查 CUDA 错误的宏
#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, const char* func, const char* file, int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        exit(1);
    }
}

// CUDA 核函数：使用 Tensor Core 进行矩阵乘法 D = A * B + C
__global__ void wmma_matrix_mult(half* a, half* b, float* c, float* d, int M, int N, int K) {
    // 声明 WMMA 碎片
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::row_major> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> acc_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> c_frag;

    // 初始化累加器碎片为 0
    nvcuda::wmma::fill_fragment(acc_frag, 0.0f);

    // 加载矩阵 A 和 B 到碎片
    nvcuda::wmma::load_matrix_sync(a_frag, a, 16); // 步长为 16
    nvcuda::wmma::load_matrix_sync(b_frag, b, 16); // 步长为 16

    // 执行矩阵乘法：acc = A * B
    nvcuda::wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

    // 加载矩阵 C 到碎片
    nvcuda::wmma::load_matrix_sync(c_frag, c, 16, nvcuda::wmma::mem_row_major);

    // 累加 C 到 acc：acc += C
    for (int i = 0; i < acc_frag.num_elements; i++) {
        acc_frag.x[i] += c_frag.x[i];
    }

    // 存储结果到 D
    nvcuda::wmma::store_matrix_sync(d, acc_frag, 16, nvcuda::wmma::mem_row_major);
}

// 主机函数：初始化矩阵并调用核函数
int main() {
    // 矩阵尺寸
    const int M = 16;
    const int N = 16;
    const int K = 16;

    // 主机端矩阵
    float* h_A = new float[M * K];
    float* h_B = new float[K * N];
    float* h_C = new float[M * N];
    float* h_D = new float[M * N];

    // 初始化矩阵 A、B、C
    for (int i = 0; i < M * K; i++) h_A[i] = 1.0f;
    for (int i = 0; i < K * N; i++) h_B[i] = 1.0f;
    for (int i = 0; i < M * N; i++) h_C[i] = 0.5f;

    // 转换为 FP16
    half* h_A_fp16 = new half[M * K];
    half* h_B_fp16 = new half[K * N];
    for (int i = 0; i < M * K; i++) h_A_fp16[i] = __float2half(h_A[i]);
    for (int i = 0; i < K * N; i++) h_B_fp16[i] = __float2half(h_B[i]);

    // 设备端指针
    half *d_A, *d_B;
    float *d_C, *d_D;
    CHECK_CUDA_ERROR(hipMalloc(&d_A, M * K * sizeof(half)));
    CHECK_CUDA_ERROR(hipMalloc(&d_B, K * N * sizeof(half)));
    CHECK_CUDA_ERROR(hipMalloc(&d_C, M * N * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_D, M * N * sizeof(float)));

    // 复制数据到设备
    CHECK_CUDA_ERROR(hipMemcpy(d_A, h_A_fp16, M * K * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, h_B_fp16, K * N * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice));

    // 设置线程块和网格
    dim3 threadsPerBlock(32, 1); // 一个 warp 有 32 个线程
    dim3 blocksPerGrid(1, 1);    // 单块处理 16x16 矩阵

    // 调用核函数
    wmma_matrix_mult<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, d_D, M, N, K);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // 复制结果回主机
    CHECK_CUDA_ERROR(hipMemcpy(h_D, d_D, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // 打印结果
    printf("Result matrix D (first 4 elements):\n");
    for (int i = 0; i < 4; i++) {
        printf("%f ", h_D[i]);
    }
    printf("\n");

    // 释放内存
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_D;
    delete[] h_A_fp16;
    delete[] h_B_fp16;
    CHECK_CUDA_ERROR(hipFree(d_A));
    CHECK_CUDA_ERROR(hipFree(d_B));
    CHECK_CUDA_ERROR(hipFree(d_C));
    CHECK_CUDA_ERROR(hipFree(d_D));

    return 0;
}